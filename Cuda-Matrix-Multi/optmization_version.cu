#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include"time.h"
#include"stdlib.h"
#include<hipblas.h>
#include<hipblas.h>

#include<iomanip>
#include<iostream>
using namespace std;

#define NUM_THREADS 256  

float *a, *b, *c, *d;
float *ag, *bg, *cg, *dg;


clock_t start, finish,start1,finish1;

double duration,duration1;

int row1, column1, row2, column2;

void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);

	if (count == 0)
	{
		fprintf(stderr, "There is no device.\n");

		return false;
	}

	int i;

	for (i = 0; i < count; i++)
	{

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
	//	printDeviceProp(prop);

		if (hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if (prop.major >= 1)
			{
				break;
			}
		}
	}
	if (i == count)
	{
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

void matrix_generate(int i,int j,int i1,int j1)
{
	srand(time(NULL));
	for (int ii = 0; ii < i; ii++)
	{
		for (int jj = 0; jj < j; jj++)
		{
			a[ii*i + jj] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);
				
		}
	}

//	srand(time(NULL));
	for (int ii = 0; ii < i1; ii++)
	{
		for (int jj = 0; jj < j1; jj++)
		{
			b[ii*i + jj] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);
		}
	}

}

void print_matrix(int row,int column)
{
	cout << "The matrix A is:" << endl;
	int i, j;
	for (i = 0; i < row; i++)
	{
		for (j = 0; j < column; j++)
			cout << setw(10) << a[i*row+j];                //Print the elements of the matrix
		cout << endl;
	}
	cout << endl;

	cout << "The matrix B is:" << endl;
	for (i = 0; i < row; i++)
	{
		for (j = 0; j < column; j++)
			cout << setw(10) << b[i*row + j];                //Print the elements of the matrix
		cout << endl;
	}
	cout << endl;

	cout << "The result CPU is:" << endl;
	for (i = 0; i < row; i++)
	{
		for (j = 0; j < column; j++)
			cout << setw(10)<<c[i*row + j];                //Print the elements of the matrix
		cout << endl;
	}
	    cout << endl;

}

void printgpu(int row, int column)
{
	cout << "The result GPU is:" << endl;
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < column; j++)
			cout << setw(10) << d[i*row + j];                //Print the elements of the matrix
		cout << endl;
	}
	cout << endl;
}

void matrixmul_cpu(int ii, int jj, int ii1, int jj1)
{

	int i, j, q;
		for (i = 0; i < ii; i++)
		{
			for (j = 0; j < jj1; j++)
			{
				c[i*jj+j] = 0;
				for (q = 0; q < ii1; q++)
				{
					c[i*jj + j] = c[i*jj + j] + a[i*jj+q] * b[j+q*ii1];
				}
			}
		}
}



__global__ static void CUDAshare(const float *aa, const float *bb,  float *cc, int n)
{
	extern __shared__ float data[];
	const int tid = threadIdx.x;
	const int row= blockIdx.x;
	int i, j;

	for (i = tid; i< n; i+= blockDim.x) 
	{
	//	printf("%f\n", data[i]);
		data[i] = aa[row*n + i];
		data[i] =5;
		//printf("%f\n", aa[row*n + i]);
		//printf("%f\n",data[i]);
	}



	__syncthreads();

	for (j = tid; j < n; j+= blockDim.x) {
		float t = 0;
		float y= 0;
		for (i  = 0; i < n; i++) {
			float r;

		//	printf("%f\n", bb[i * n + j]);
		//	printf("%f\n", data[i]);

			y-= data[i] * bb[i * n  + j];
			r= t - y;
			y= (r - t) + y;
			t = r;
		}
		cc[row*n + j] = t;
	}
}


__global__ void matrixgpu(float *aa, float *bb, float *cc, int n)
{
	
	
	int index = blockIdx.x *blockDim.x + threadIdx.x;
	int row = index / n;
	int column = index%n;
	if (row < n && column < n)
	{
		float t = 0;

		for (int ii = 0; ii < n; ii=ii+5)
		{
			t = t + aa[row*n + ii] * bb[ii*n + column];
			t = t + aa[row*n + ii + 1] * bb[(ii + 1)*n + column];
			t = t + aa[row*n + ii + 2] * bb[(ii + 2)*n + column];
			t = t + aa[row*n + ii + 3] * bb[(ii + 3)*n + column];
			t = t + aa[row*n + ii + 4] * bb[(ii + 4)*n + column];

		}
		cc[row*n + column] = t;
	}

}



int main()
{
	int block;
	cout<<"Input the dimension matrix"<<endl;
	cin >> row1;

	cout << "The dimension of matrixes: " << row1 << endl;

	if (InitCUDA() == false)
	{
		return 0;
	}

	a = (float*)malloc(sizeof(float)* row1 * row1);
	b = (float*)malloc(sizeof(float)* row1 * row1);
	c = (float*)malloc(sizeof(float)* row1 * row1);
	d = (float*)malloc(sizeof(float)* row1 * row1);

	hipMalloc((void**)&ag, sizeof(float)*row1*row1);
	hipMalloc((void**)&bg, sizeof(float)*row1*row1);
	hipMalloc((void**)&cg, sizeof(float)*row1*row1);
	
	matrix_generate(row1,row1,row1,row1 );

	
	  hipMemcpy(ag, a, sizeof(float)*row1*row1, hipMemcpyHostToDevice);
	  hipMemcpy(bg, b, sizeof(float)*row1*row1, hipMemcpyHostToDevice);

	start = clock();

	matrixmul_cpu(row1, row1, row1, row1);
	finish = clock();
	duration = (double)(finish - start) /( CLOCKS_PER_SEC/1000);

	cout << "The CPU time is " << duration<<" ms"<<endl;

	block = 1+row1*row1 / NUM_THREADS;

	start1 = clock();
	
	//CUDAshare <<<block, NUM_THREADS >>>(ag,bg, cg, row1);
	matrixgpu << <block, NUM_THREADS >> >(ag, bg, cg, row1);

	hipDeviceSynchronize();

	finish1 = clock();
	
	hipMemcpy(d, cg, sizeof(float)* row1*row1, hipMemcpyDeviceToHost);

	duration1 = (double)(finish1 - start1) / (CLOCKS_PER_SEC / 1000);

	cout << "The GPU time is " << duration1 << " ms" << endl;


	print_matrix(row1, row1);

	printgpu(row1, row1);
	hipFree(ag);
	hipFree(bg);
	hipFree(cg);

    return 0;
}


